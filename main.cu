#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/shuffle.h>

#include "cub/device/device_partition.cuh"
#include "cub/iterator/cache_modified_input_iterator.cuh"
#include "cub/iterator/cache_modified_output_iterator.cuh"
#include "cub/device/device_batch_memcpy.cuh"
#include "cub/block/block_load.cuh"
#include "cub/block/block_store.cuh"
#include "cub/warp/warp_load.cuh"
#include "cub/warp/warp_store.cuh"


float get_max_bw(int dev = 0)
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);

  return float(deviceProp.memoryBusWidth) * deviceProp.memoryClockRate * 2 / 8 / 1000 / 1000;
}


template <typename T>
thrust::host_vector<T> gen_uniform_buffer_sizes(std::size_t buffers,
                                                std::size_t buffer_size)
{
  thrust::host_vector<T> sizes(buffers, buffer_size);
  return sizes;
}

template <typename T>
thrust::host_vector<T> gen_shuffled_buffer_sizes(std::size_t small_buffers,
                                                 std::size_t medium_buffers,
                                                 std::size_t large_buffers,
                                                 std::size_t small_buffer_size,
                                                 std::size_t medium_buffer_size,
                                                 std::size_t large_buffer_size)
{
  const std::size_t total_buffers = small_buffers + medium_buffers + large_buffers;
  thrust::host_vector<T> sizes(total_buffers);

  thrust::fill_n(sizes.begin(), small_buffers, small_buffer_size);
  thrust::fill_n(sizes.begin() + small_buffers, medium_buffers, medium_buffer_size);
  thrust::fill_n(sizes.begin() + small_buffers + medium_buffers, large_buffers, large_buffer_size);

  thrust::default_random_engine re;
  thrust::shuffle(sizes.begin(), sizes.end(), re);

  return sizes;
}

template <typename DataT, typename OffsetT>
class Input
{
  const std::size_t buffers {};
  std::size_t total_input_size {};

  mutable thrust::device_vector<DataT> input;
  mutable thrust::device_vector<DataT> output;

  thrust::device_vector<void*> in_pointers;
  thrust::device_vector<void*> out_pointers;

  thrust::device_vector<OffsetT> buffer_sizes;

public:
  Input(thrust::host_vector<OffsetT> h_buffer_sizes)
    : buffers(h_buffer_sizes.size())
    , in_pointers(buffers)
    , out_pointers(buffers)
  {
    total_input_size = thrust::reduce(h_buffer_sizes.begin(),
                                      h_buffer_sizes.end());

    input.resize(total_input_size);
    output.resize(total_input_size);

    thrust::host_vector<void*> h_in_pointers(buffers);
    thrust::host_vector<void*> h_out_pointers(buffers);

    DataT *in_ptr = thrust::raw_pointer_cast(input.data());
    DataT *out_ptr = thrust::raw_pointer_cast(output.data());

    for (std::size_t buffer = 0; buffer < buffers; buffer++)
    {
      h_in_pointers[buffer] = in_ptr;
      h_out_pointers[buffer] = out_ptr;

      in_ptr += h_buffer_sizes[buffer];
      out_ptr += h_buffer_sizes[buffer];

      h_buffer_sizes[buffer] *= sizeof(DataT);
    }

    in_pointers = h_in_pointers;
    out_pointers = h_out_pointers;
    buffer_sizes = h_buffer_sizes;
  }

  void fill_input(DataT value) const
  {
    thrust::fill(input.begin(), input.end(), value);
  }

  void fill_output(DataT value) const
  {
    thrust::fill(output.begin(), output.end(), value);
  }

  void compare() const
  {
    if (output != input)
    {
      throw std::runtime_error("Wrong result!");
    }
  }

  std::size_t get_bytes_read() const
  {
    return total_input_size * sizeof(DataT);
  }

  std::size_t get_bytes_written() const
  {
    return get_bytes_read();
  }

  void** get_input() const
  {
    return const_cast<void**>(thrust::raw_pointer_cast(in_pointers.data()));
  }

  void** get_output() const
  {
    return const_cast<void**>(thrust::raw_pointer_cast(out_pointers.data()));
  }

  void* get_input_raw() const
  {
    return const_cast<DataT*>(thrust::raw_pointer_cast(input.data()));
  }

  void* get_output_raw() const
  {
    return const_cast<DataT*>(thrust::raw_pointer_cast(output.data()));
  }

  const OffsetT* get_buffer_sizes() const
  {
    return thrust::raw_pointer_cast(buffer_sizes.data());
  }

  std::size_t get_num_buffers() const
  {
    return buffers;
  }

  float bytes_to_gb(std::size_t bytes) const
  {
    return static_cast<float>(bytes) / 1024.0f / 1024.0f / 1024.0f;
  }

  float get_bw(float ms) const
  {
    float seconds = ms / 1000.0f;
    return bytes_to_gb(get_bytes_read() + get_bytes_written()) / seconds;
  }
};


template <typename DataT,
          typename OffsetT>
void report_result(float ms, const Input<DataT, OffsetT> &input)
{
  const float achieved_bw = input.get_bw(ms);
  const float expected_bw = get_max_bw();

  std::cout << achieved_bw << " / " << expected_bw << " ("
            << (achieved_bw / expected_bw) * 100.0f << "%)" << std::endl;
}


template <typename DataT,
          typename OffsetT>
void measure_cub(const Input<DataT, OffsetT> &input)
{
  std::size_t temp_storage_bytes {};
  hipcub::DeviceBatchMemcpy(nullptr,
                         temp_storage_bytes,
                         input.get_input(),
                         input.get_output(),
                         input.get_buffer_sizes(),
                         input.get_num_buffers());

  thrust::device_vector<std::uint8_t> temp_storage(temp_storage_bytes);
  std::uint8_t *d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

  input.fill_input(DataT{42});
  input.fill_output(DataT{1});

  hipEvent_t begin, end;
  hipEventCreate(&begin);
  hipEventCreate(&end);

  hipEventRecord(begin);

  hipcub::DeviceBatchMemcpy(d_temp_storage,
                         temp_storage_bytes,
                         input.get_input(),
                         input.get_output(),
                         input.get_buffer_sizes(),
                         input.get_num_buffers());

  hipEventRecord(end);
  hipEventSynchronize(end);

  float ms {};
  hipEventElapsedTime(&ms, begin, end);

  input.compare();

  report_result(ms, input);

  hipEventDestroy(end);
  hipEventDestroy(begin);
}


template <int BlockThreads,
          typename OffsetT>
__launch_bounds__(BlockThreads)
__global__ void naive_kernel(void **in_pointers,
                             void **out_pointers,
                             const OffsetT *sizes)
{
  using underlying_type = std::uint32_t;

  constexpr int items_per_thread = 4;
  constexpr int tile_size = items_per_thread * BlockThreads;

  using BlockLoadT =
    hipcub::BlockLoad<underlying_type,
                   BlockThreads,
                   items_per_thread,
                   hipcub::BlockLoadAlgorithm::BLOCK_LOAD_VECTORIZE>;

  using BlockStoreT =
    hipcub::BlockStore<underlying_type,
                    BlockThreads,
                    items_per_thread,
                    hipcub::BlockStoreAlgorithm::BLOCK_STORE_VECTORIZE>;

  __shared__ union
  {
    typename BlockLoadT::TempStorage load;
    typename BlockStoreT::TempStorage store;

  } storage;

  const int buffer_id = blockIdx.x;
  auto in = reinterpret_cast<underlying_type*>(in_pointers[buffer_id]);
  auto out = reinterpret_cast<underlying_type*>(out_pointers[buffer_id]);
  const auto size = sizes[buffer_id];
  const auto size_in_elements = size / sizeof(underlying_type);
  const auto tiles = size_in_elements / tile_size;

  for (std::size_t tile = 0; tile < tiles; tile++)
  {
    hipcub::CacheModifiedInputIterator<hipcub::CacheLoadModifier::LOAD_CS, underlying_type> in_iterator(in);
    hipcub::CacheModifiedOutputIterator<hipcub::CacheStoreModifier::STORE_CS, underlying_type> out_iterator(out);

    underlying_type thread_data[items_per_thread];
    BlockLoadT(storage.load).Load(in_iterator, thread_data);
    BlockStoreT(storage.store).Store(out_iterator, thread_data);

    in += tile_size;
    out += tile_size;
  }
}


template <typename DataT,
          typename OffsetT>
void measure_naive(const Input<DataT, OffsetT> &input)
{
  hipEvent_t begin, end;
  hipEventCreate(&begin);
  hipEventCreate(&end);

  input.fill_input(DataT{24});
  input.fill_output(DataT{1});

  hipEventRecord(begin);

  constexpr int block_threads = 256;
  naive_kernel<block_threads, OffsetT>
    <<<input.get_num_buffers(), block_threads>>>(
      input.get_input(),
      input.get_output(),
      input.get_buffer_sizes());

  hipEventRecord(end);
  hipEventSynchronize(end);

  float ms {};
  hipEventElapsedTime(&ms, begin, end);

  input.compare();

  report_result(ms, input);

  hipEventDestroy(end);
  hipEventDestroy(begin);
}


template <int BlockThreads,
          typename OffsetT>
__launch_bounds__(BlockThreads)
__global__ void large_kernel(
    int large_buffers,
    const int *large_buffers_reordering,
    int *tiles_copied_ptr,

    void **in_pointers,
    void **out_pointers,
    const OffsetT *sizes)
{
  using underlying_type = std::uint32_t;

  constexpr int items_per_thread = 4;
  constexpr int tile_size = items_per_thread * BlockThreads;
  constexpr int tiles_per_request = 2;

  using BlockLoadT =
    hipcub::BlockLoad<underlying_type,
                   BlockThreads,
                   items_per_thread,
                   hipcub::BlockLoadAlgorithm::BLOCK_LOAD_VECTORIZE>;

  using BlockStoreT =
    hipcub::BlockStore<underlying_type,
                    BlockThreads,
                    items_per_thread,
                    hipcub::BlockStoreAlgorithm::BLOCK_STORE_VECTORIZE>;

  __shared__ union
  {
    typename BlockLoadT::TempStorage load;
    typename BlockStoreT::TempStorage store;

  } storage;

  for (unsigned int bid = blockIdx.x % large_buffers; bid < large_buffers; bid += gridDim.x)
  {
    const int buffer_id = large_buffers_reordering[bid];

    auto in_origin =
      reinterpret_cast<underlying_type *>(in_pointers[buffer_id]);
    auto out_origin =
      reinterpret_cast<underlying_type *>(out_pointers[buffer_id]);
    const auto size             = sizes[buffer_id];
    const auto size_in_elements = size / sizeof(underlying_type);
    const auto tiles            = size_in_elements / tile_size;

    __shared__ int tiles_copied_cache;

    if (threadIdx.x == 0)
    {
      tiles_copied_cache = atomicAdd(tiles_copied_ptr + buffer_id,
                                     tiles_per_request);
    }
    __syncthreads();
    int tiles_copied = tiles_copied_cache;

    while (tiles_copied < tiles)
    {
      for (std::size_t tile = 0; tile < tiles_per_request; tile++)
      {
        if (tile + tiles_copied >= tiles)
        {
          break;
        }

        const OffsetT tile_offset = (tile + tiles_copied) * tile_size;
        const auto in             = in_origin + tile_offset;
        const auto out            = out_origin + tile_offset;

        hipcub::CacheModifiedInputIterator<hipcub::CacheLoadModifier::LOAD_CS,
                                        underlying_type>
          in_iterator(in);
        hipcub::CacheModifiedOutputIterator<hipcub::CacheStoreModifier::STORE_CS,
                                         underlying_type>
          out_iterator(out);

        underlying_type thread_data[items_per_thread];
        BlockLoadT(storage.load).Load(in_iterator, thread_data);
        BlockStoreT(storage.store).Store(out_iterator, thread_data);
      }

      if (threadIdx.x == 0)
      {
        tiles_copied_cache = atomicAdd(tiles_copied_ptr + buffer_id,
                                       tiles_per_request);
      }
      __syncthreads();
      tiles_copied = tiles_copied_cache;
    }
  }
}


template <typename DataT,
  typename OffsetT>
void measure_large(const Input<DataT, OffsetT> &input)
{
  hipEvent_t begin, end;
  hipEventCreate(&begin);
  hipEventCreate(&end);


  thrust::device_vector<int> buffers_reordering(input.get_num_buffers());
  thrust::sequence(buffers_reordering.begin(), buffers_reordering.end());
  const int *d_buffers_reordering = thrust::raw_pointer_cast(buffers_reordering.data());

  thrust::device_vector<int> tiles_copied(input.get_num_buffers());
  int *d_tiles_copied = thrust::raw_pointer_cast(tiles_copied.data());

  input.fill_input(DataT{24});
  input.fill_output(DataT{1});


  constexpr int block_threads = 256;


  int sm_count;
  int dev_id = 0;
  hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev_id);

  // Get SM occupancy for the batch memcpy block-level buffers kernel
  int max_occupancy;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(
    &max_occupancy,
    large_kernel<block_threads, OffsetT>,
    block_threads,
    0);

  const int grid_size = max_occupancy * sm_count;


  hipEventRecord(begin);

  large_kernel<block_threads, OffsetT>
    <<<grid_size, block_threads>>>(

      input.get_num_buffers(),
      d_buffers_reordering,
      d_tiles_copied,

      input.get_input(),
      input.get_output(),
      input.get_buffer_sizes());

  hipEventRecord(end);
  hipEventSynchronize(end);

  float ms{};
  hipEventElapsedTime(&ms, begin, end);

  input.compare();

  report_result(ms, input);

  hipEventDestroy(end);
  hipEventDestroy(begin);
}


template <typename DataT,
          typename OffsetT>
void measure_memcpy(const Input<DataT, OffsetT> &input)
{
  hipEvent_t begin, end;
  hipEventCreate(&begin);
  hipEventCreate(&end);


  input.fill_input(DataT{24});
  input.fill_output(DataT{1});

  hipEventRecord(begin);

  hipMemcpyAsync(input.get_output_raw(),
                  input.get_input_raw(),
                  input.get_bytes_written(),
                  hipMemcpyDeviceToDevice);

  hipEventRecord(end);
  hipEventSynchronize(end);

  float ms{};
  hipEventElapsedTime(&ms, begin, end);

  input.compare();

  report_result(ms, input);

  hipEventDestroy(end);
  hipEventDestroy(begin);
}


template <typename OffsetT>
struct LargeSegmentsSelectorT
{
  OffsetT value{};
  const OffsetT *d_sizes{};

  __host__ __device__ __forceinline__
  LargeSegmentsSelectorT(OffsetT value,
                         const OffsetT *d_sizes)
    : value(value)
    , d_sizes(d_sizes)
  {}

  __host__ __device__ __forceinline__ bool
  operator()(int segment_id) const
  {
    return d_sizes[segment_id] > value;
  }
};


template <typename OffsetT>
struct SmallSegmentsSelectorT
{
  OffsetT value{};
  const OffsetT *d_sizes{};

  __host__ __device__ __forceinline__
  SmallSegmentsSelectorT(OffsetT value,
                         const OffsetT *d_sizes)
    : value(value)
    , d_sizes(d_sizes)
  {}

  __host__ __device__ __forceinline__ bool
  operator()(unsigned int segment_id) const
  {
    return d_sizes[segment_id] < value;
  }
};

template <int BlockThreads,
          typename LargeBuffersReorderingT,
          typename MediumBuffersReorderingT,
          typename OffsetT>
__launch_bounds__(BlockThreads)
__global__ void partitioned_kernel(
  int num_buffers,
  const int *d_group_sizes,
  LargeBuffersReorderingT large_buffers_reordering,
  MediumBuffersReorderingT medium_buffers_reordering,
  int *small_buffers_reordering,
  int *tiles_copied_ptr,

  void **in_pointers,
  void **out_pointers,
  const OffsetT *sizes)
{
  using underlying_type = std::uint32_t;

  const unsigned int large_buffers = d_group_sizes[0];
  const unsigned int small_buffers = d_group_sizes[1];
  const unsigned int medium_buffers = num_buffers - large_buffers - small_buffers;

  constexpr int items_per_thread  = 4;
  constexpr int tile_size         = items_per_thread * BlockThreads;
  constexpr int tiles_per_request = 2;

  using BlockLoadT =
    hipcub::BlockLoad<underlying_type,
                   BlockThreads,
                   items_per_thread,
                   hipcub::BlockLoadAlgorithm::BLOCK_LOAD_VECTORIZE>;

  using BlockStoreT =
    hipcub::BlockStore<underlying_type,
                    BlockThreads,
                    items_per_thread,
                    hipcub::BlockStoreAlgorithm::BLOCK_STORE_VECTORIZE>;

  constexpr int warp_size = 4;

  /*
   *
template <typename          InputT,
          int               ITEMS_PER_THREAD,
          WarpLoadAlgorithm ALGORITHM            = WARP_LOAD_DIRECT,
          int               LOGICAL_WARP_THREADS = HIPCUB_WARP_THREADS,
   */

  using WarpLoadT = hipcub::WarpLoad<underlying_type,
                                  items_per_thread,
                                  hipcub::WarpLoadAlgorithm::WARP_LOAD_VECTORIZE,
                                  warp_size>;
  using WarpStoreT =
    hipcub::WarpStore<underlying_type,
                   items_per_thread,
                   hipcub::WarpStoreAlgorithm::WARP_STORE_VECTORIZE,
                   warp_size>;

  __shared__ union
  {
    typename BlockLoadT::TempStorage block_load;
    typename BlockStoreT::TempStorage block_store;

    typename WarpLoadT::TempStorage warp_load;
    typename WarpStoreT::TempStorage warp_store;
  } storage;


  for (unsigned int bid = blockIdx.x; bid < medium_buffers; bid += gridDim.x)
  {
    const int buffer_id = medium_buffers_reordering[bid];

    auto in = reinterpret_cast<underlying_type *>(in_pointers[buffer_id]);
    auto out = reinterpret_cast<underlying_type *>(out_pointers[buffer_id]);
    const auto size             = sizes[buffer_id];
    const auto size_in_elements = size / sizeof(underlying_type);
    const auto tiles            = size_in_elements / tile_size;

    for (std::size_t tile = 0; tile < tiles; tile++)
    {
      hipcub::CacheModifiedInputIterator<hipcub::CacheLoadModifier::LOAD_CS, underlying_type> in_iterator(in);
      hipcub::CacheModifiedOutputIterator<hipcub::CacheStoreModifier::STORE_CS, underlying_type> out_iterator(out);

      underlying_type thread_data[items_per_thread];
      BlockLoadT(storage.block_load).Load(in_iterator, thread_data);
      BlockStoreT(storage.block_store).Store(out_iterator, thread_data);

      in += tile_size;
      out += tile_size;
    }
  }

  if (large_buffers > 0)
  {
    for (unsigned int bid = blockIdx.x % large_buffers; bid < large_buffers; bid += gridDim.x)
    {
      const int buffer_id = large_buffers_reordering[bid];

      auto in_origin = reinterpret_cast<underlying_type *>(in_pointers[buffer_id]);
      auto out_origin = reinterpret_cast<underlying_type *>(out_pointers[buffer_id]);
      const auto size             = sizes[buffer_id];
      const auto size_in_elements = size / sizeof(underlying_type);
      const auto tiles            = (size_in_elements + tile_size - 1) / tile_size;

      __shared__ int tiles_copied_cache;

      if (threadIdx.x == 0)
      {
        tiles_copied_cache = atomicAdd(tiles_copied_ptr + buffer_id,
                                       tiles_per_request);
      }
      __syncthreads();
      int tiles_copied = tiles_copied_cache;

      bool process_last_tile = false;

      while (tiles_copied < tiles)
      {
        for (std::size_t tile = 0; tile < tiles_per_request; tile++)
        {
          if (tile + tiles_copied == tiles - 1)
          {
            process_last_tile = true;
            break;
          }

          if (tile + tiles_copied >= tiles)
          {
            break;
          }

          const OffsetT tile_offset = (tile + tiles_copied) * tile_size;
          const auto in             = in_origin + tile_offset;
          const auto out            = out_origin + tile_offset;

          hipcub::CacheModifiedInputIterator<hipcub::CacheLoadModifier::LOAD_CS, underlying_type> in_iterator(in);
          hipcub::CacheModifiedOutputIterator<hipcub::CacheStoreModifier::STORE_CS, underlying_type> out_iterator(out);

          underlying_type thread_data[items_per_thread];
          BlockLoadT(storage.block_load).Load(in_iterator, thread_data);
          BlockStoreT(storage.block_store).Store(out_iterator, thread_data);
        }

        if (process_last_tile)
        {
          const OffsetT tile_offset = (tiles - 1) * tile_size;
          const auto in = in_origin + tile_offset;
          const auto out = out_origin + tile_offset;

          const int valid_items = size_in_elements - (tiles - 1) * tile_size;

          hipcub::CacheModifiedInputIterator<hipcub::CacheLoadModifier::LOAD_CS, underlying_type> in_iterator(in);
          hipcub::CacheModifiedOutputIterator<hipcub::CacheStoreModifier::STORE_CS, underlying_type> out_iterator(out);

          underlying_type thread_data[items_per_thread];
          BlockLoadT(storage.block_load).Load(in_iterator, thread_data, valid_items);
          BlockStoreT(storage.block_store).Store(out_iterator, thread_data, valid_items);
          break;
        }
        else
        {
          if (threadIdx.x == 0)
          {
            tiles_copied_cache = atomicAdd(tiles_copied_ptr + buffer_id,
                                           tiles_per_request);
          }
          __syncthreads();
          tiles_copied = tiles_copied_cache;
        }
      }
    }
  }

  constexpr unsigned int warp_tile_size = warp_size * items_per_thread;
  const unsigned int warp_id = (BlockThreads * blockIdx.x + threadIdx.x) / warp_size;
  const unsigned int total_warps = (BlockThreads * gridDim.x) / warp_size;

  for (unsigned int bid = warp_id; bid < small_buffers; bid += total_warps)
  {
    const int buffer_id = small_buffers_reordering[bid];

    auto in = reinterpret_cast<underlying_type *>(in_pointers[buffer_id]);
    auto out = reinterpret_cast<underlying_type *>(out_pointers[buffer_id]);
    const auto size             = sizes[buffer_id];
    const auto size_in_elements = size / sizeof(underlying_type);
    const auto tiles            = size_in_elements / warp_tile_size;

    for (std::size_t tile = 0; tile < tiles; tile++)
    {
      hipcub::CacheModifiedInputIterator<hipcub::CacheLoadModifier::LOAD_CS, underlying_type> in_iterator(in);
      hipcub::CacheModifiedOutputIterator<hipcub::CacheStoreModifier::STORE_CS, underlying_type> out_iterator(out);

      underlying_type thread_data[items_per_thread];
      WarpLoadT(storage.warp_load).Load(in_iterator, thread_data);
      WarpStoreT(storage.warp_store).Store(out_iterator, thread_data);

      in += warp_tile_size;
      out += warp_tile_size;
    }
  }
}


template <typename DataT,
          typename OffsetT>
void measure_partition(const Input<DataT, OffsetT> &input)
{
  const std::size_t num_buffers = input.get_num_buffers();

  thrust::counting_iterator<int> buffer_ids(0);

  thrust::device_vector<int> small_and_large(num_buffers);
  thrust::device_vector<int> queue_and_medium(num_buffers);

  int *d_small_and_large = thrust::raw_pointer_cast(small_and_large.data());
  int *d_queue_and_medium = thrust::raw_pointer_cast(queue_and_medium.data());

  auto medium = thrust::make_reverse_iterator(d_queue_and_medium + num_buffers);
  auto large = thrust::make_reverse_iterator(d_small_and_large + num_buffers);

  const OffsetT *d_sizes = input.get_buffer_sizes();

  const OffsetT small_segment_max_size = 1024; // bytes
  SmallSegmentsSelectorT<OffsetT> small_selector(small_segment_max_size, d_sizes);

  const OffsetT large_segment_min_size = 1024 * 1024; // bytes
  LargeSegmentsSelectorT<OffsetT> large_selector(large_segment_min_size, d_sizes);

  thrust::device_vector<int> group_sizes(2);
  int *d_group_sizes = thrust::raw_pointer_cast(group_sizes.data());


  std::size_t temp_storage_bytes;
  hipcub::DevicePartition::If(nullptr,
                           temp_storage_bytes,
                           buffer_ids,
                           large,
                           d_small_and_large,
                           medium,
                           d_group_sizes,
                           num_buffers,
                           small_selector,
                           large_selector);

  thrust::device_vector<std::uint8_t> temp_storage(temp_storage_bytes);
  std::uint8_t *d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());


  hipEvent_t begin, end;
  hipEventCreate(&begin);
  hipEventCreate(&end);


  input.fill_input(DataT{35});
  input.fill_output(DataT{2});

  hipEventRecord(begin);

  constexpr int block_threads = 256;

  int dev_id = 0;
  int sm_count;
  hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev_id);

  // Get SM occupancy for the batch memcpy block-level buffers kernel
  int max_occupancy;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(
    &max_occupancy,
    partitioned_kernel<block_threads, decltype(large), decltype(medium), OffsetT>,
    block_threads,
    0);

  const int grid_size = max_occupancy * sm_count;

  hipMemsetAsync(d_queue_and_medium, 0, sizeof(int) * num_buffers);

  hipcub::DevicePartition::If(d_temp_storage,
                           temp_storage_bytes,
                           buffer_ids,
                           large,
                           d_small_and_large,
                           medium,
                           d_group_sizes,
                           num_buffers,
                           large_selector,
                           small_selector);

  partitioned_kernel<block_threads>
    <<<grid_size, block_threads>>>(num_buffers,
                                   d_group_sizes,
                                   large,
                                   medium,
                                   d_small_and_large,
                                   d_queue_and_medium,

                                   input.get_input(),
                                   input.get_output(),
                                   input.get_buffer_sizes());

  hipEventRecord(end);
  hipEventSynchronize(end);

  float ms{};
  hipEventElapsedTime(&ms, begin, end);

  input.compare();

  report_result(ms, input);

  hipEventDestroy(end);
  hipEventDestroy(begin);
}


int main()
{
  const auto input = Input<std::uint32_t, std::uint32_t>(
    gen_shuffled_buffer_sizes<std::uint32_t>(
      0, // small
      0, // medium
      300, // large,
      96,
      256 * 4 * 16,
      2 * 1024 * 1024));

  // 1024 * 1024 buffers of 256 elements => 46%
  // 1024 buffers of 1024 * 1024 elements => 78%
  // 2 buffers of 256 * 1024 * 1024 elements => 79%
  measure_cub(input);
  // measure_naive(input);
  // measure_large(input);
  measure_memcpy(input);

  measure_partition(input);

  return 0;
}
